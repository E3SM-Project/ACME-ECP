
#include "BuddyAllocator.h"

BuddyAllocator ba;


extern "C" void gatorInit( size_t bytes ) {
  auto myalloc = [] (size_t bytes) { void* ptr; hipMallocManaged(&ptr,bytes); return ptr; };
  auto myfree  = [] (void* ptr) {hipFree(ptr);};
  ba = BuddyAllocator( bytes , 1024 , myalloc , myfree );
}


extern "C" void gatorFinalize( ) {
  ba = BuddyAllocator();
}


extern "C" void* gatorAllocate( size_t bytes ) {
  return ba.allocate( bytes );
}


extern "C" void gatorDeallocate( void *ptr ) {
  ba.free( ptr );
}


